#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <time.h>
#include <string>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#define THREADS_PER_BLOCK 512

using namespace std;

int SBoxInvert(int num) {

	int rsbox[256] =

	{ 0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb

		, 0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb

		, 0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e

		, 0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25

		, 0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92

		, 0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84

		, 0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06

		, 0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b

		, 0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73

		, 0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e

		, 0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b

		, 0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4

		, 0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f

		, 0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef

		, 0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61

		, 0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d };



	return rsbox[num];

}

int SBoxValue(int num) {

	int sbox[256] = {

		0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,

		0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,

		0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,

		0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,

		0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,

		0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,

		0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,

		0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,

		0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,

		0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,

		0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,

		0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,

		0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,

		0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,

		0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,

		0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };

	return sbox[num];

}

void subBytes(unsigned char **state) {
	for (int i = 0; i<4; i++)
		for (int j = 0; j<4; j++)
			state[i][j] = SBoxValue(state[i][j]);
}

void invSubBytes(unsigned char **state) {
	for (int i = 0; i<4; i++)
		for (int j = 0; j<4; j++)
			state[i][j] = SBoxInvert(state[i][j]);
}

void addRoundKey(int round, unsigned char **state, unsigned char *roundKey) {
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < 4; j++)
			state[j][i] ^= roundKey[round * 16 + i * 4 + j];
}

void invShiftRows(unsigned char **state)

{

	unsigned char temp;

	temp = state[1][3];

	state[1][3] = state[1][2];
	state[1][2] = state[1][1];
	state[1][1] = state[1][0];
	state[1][0] = temp;


	temp = state[2][0];

	state[2][0] = state[2][2];
	state[2][2] = temp;

	temp = state[2][1];

	state[2][1] = state[2][3];
	state[2][3] = temp;


	temp = state[3][0];

	state[3][0] = state[3][1];
	state[3][1] = state[3][2];
	state[3][2] = state[3][3];
	state[3][3] = temp;

}

void shiftRows(unsigned char **state) {

	unsigned char temp;

	temp = state[1][0];

	state[1][0] = state[1][1];
	state[1][1] = state[1][2];
	state[1][2] = state[1][3];
	state[1][3] = temp;

	temp = state[2][0];

	state[2][0] = state[2][2];
	state[2][2] = temp;

	temp = state[2][1];

	state[2][1] = state[2][3];
	state[2][3] = temp;

	temp = state[3][3];

	state[3][3] = state[3][2];
	state[3][2] = state[3][1];
	state[3][1] = state[3][0];
	state[3][0] = temp;


}

void showState(unsigned char **state) {
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			cout << (int)state[j][i] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void keyExpansion(unsigned char *roundKey, unsigned char key[]) {

	int rcon[10] = { 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

	unsigned char temp[4], k;
	for (int i = 0; i < 4; i++) {
		roundKey[i * 4] = key[i * 4];
		roundKey[i * 4 + 1] = key[i * 4 + 1];
		roundKey[i * 4 + 2] = key[i * 4 + 2];
		roundKey[i * 4 + 3] = key[i * 4 + 3];
	}
	for (int i = 0; i < 40; i++) {
		for (int j = 0; j < 4; j++) {
			temp[j] = roundKey[i * 4 + j + 16];
		}

		if (i % 4 == 0) {

			k = temp[0];
			temp[0] = temp[1];
			temp[1] = temp[2];
			temp[2] = temp[3];
			temp[3] = k;

			temp[0] = SBoxValue(temp[0]);
			temp[1] = SBoxValue(temp[1]);
			temp[2] = SBoxValue(temp[2]);
			temp[3] = SBoxValue(temp[3]);

			temp[0] = temp[0] ^ rcon[i / 4];

			roundKey[i * 4 + 16] = roundKey[i * 4 + 0] ^ temp[0];
			roundKey[i * 4 + 17] = roundKey[i * 4 + 1] ^ temp[1];
			roundKey[i * 4 + 18] = roundKey[i * 4 + 2] ^ temp[2];
			roundKey[i * 4 + 19] = roundKey[i * 4 + 3] ^ temp[3];
		}
		else {
			roundKey[i * 4 + 16] = roundKey[i * 4 + 12] ^ roundKey[i * 4];
			roundKey[i * 4 + 17] = roundKey[i * 4 + 13] ^ roundKey[i * 4 + 1];
			roundKey[i * 4 + 18] = roundKey[i * 4 + 14] ^ roundKey[i * 4 + 2];
			roundKey[i * 4 + 19] = roundKey[i * 4 + 15] ^ roundKey[i * 4 + 3];
		}
	}
}

__device__ int dSBoxInvert(int num) {

	int rsbox[256] =

	{ 0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb

		, 0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb

		, 0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e

		, 0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25

		, 0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92

		, 0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84

		, 0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06

		, 0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b

		, 0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73

		, 0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e

		, 0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b

		, 0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4

		, 0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f

		, 0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef

		, 0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61

		, 0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d };



	return rsbox[num];

}

__device__ int dSBoxValue(int num) {

	int sbox[256] = {

		0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,

		0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,

		0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,

		0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,

		0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,

		0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,

		0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,

		0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,

		0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,

		0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,

		0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,

		0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,

		0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,

		0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,

		0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,

		0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };

	return sbox[num];

}

__device__ void dsubBytes(unsigned char **state) {
	for (int i = 0; i<4; i++)
		for (int j = 0; j<4; j++)
			state[i][j] = dSBoxValue(state[i][j]);
}

__device__ void dinvSubBytes(unsigned char **state) {
	for (int i = 0; i<4; i++)
		for (int j = 0; j<4; j++)
			state[i][j] = dSBoxInvert(state[i][j]);
}

__device__ void daddRoundKey(int round, unsigned char **state, unsigned char *roundKey) {
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < 4; j++)
			state[j][i] ^= roundKey[round * 16 + i * 4 + j];
}

__device__ void dinvShiftRows(unsigned char **state)

{

	unsigned char temp;

	temp = state[1][3];

	state[1][3] = state[1][2];
	state[1][2] = state[1][1];
	state[1][1] = state[1][0];
	state[1][0] = temp;


	temp = state[2][0];

	state[2][0] = state[2][2];
	state[2][2] = temp;

	temp = state[2][1];

	state[2][1] = state[2][3];
	state[2][3] = temp;


	temp = state[3][0];

	state[3][0] = state[3][1];
	state[3][1] = state[3][2];
	state[3][2] = state[3][3];
	state[3][3] = temp;

}

__device__ void dshiftRows(unsigned char **state) {

	unsigned char temp;

	temp = state[1][0];

	state[1][0] = state[1][1];
	state[1][1] = state[1][2];
	state[1][2] = state[1][3];
	state[1][3] = temp;

	temp = state[2][0];

	state[2][0] = state[2][2];
	state[2][2] = temp;

	temp = state[2][1];

	state[2][1] = state[2][3];
	state[2][3] = temp;

	temp = state[3][3];

	state[3][3] = state[3][2];
	state[3][2] = state[3][1];
	state[3][1] = state[3][0];
	state[3][0] = temp;


}

__global__ void dencryption(unsigned char *dataIn, unsigned char *dataOut, int *size, unsigned char *roundKey) {
	
	int size1 = *size/16;
	unsigned char input[16];
	unsigned char encryptedPartOfData[16];
	
	unsigned char **state = new unsigned char*[4];
	for (int i = 0; i < 4; i++)
		state[i] = new unsigned char[4];
		
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < size1) {
		for (int i = 0; i < 16; i++)
			input[i] = dataIn[16 * index + i];

		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++)
				state[j][i] = input[4 * i + j];

		daddRoundKey(0, state, roundKey);

		for (int i = 1; i <= 10; i++) {
			dsubBytes(state);
			dshiftRows(state);
			daddRoundKey(i, state, roundKey);
		}
		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++)
				encryptedPartOfData[4 * i + j] = state[j][i];

		for (int i = 0; i < 16; i++)
			dataOut[16 * index + i] = encryptedPartOfData[i];
	}

	for (int i = 0; i < 4; i++)
	{
		delete[]state[i];
	}
	delete[] state;
}

__global__ void ddecryption(unsigned char *dataIn, unsigned char *dataOut, int *size, unsigned char *roundKey) {

	int size1 = *size / 16;
	unsigned char input[16];
	unsigned char encryptedPartOfData[16];

	unsigned char **state = new unsigned char*[4];
	for (int i = 0; i < 4; i++)
		state[i] = new unsigned char[4];

	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < size1) {
		for (int i = 0; i < 16; i++)
			input[i] = dataIn[16 * index + i];

		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++)
				state[j][i] = input[4 * i + j];

		daddRoundKey(10, state, roundKey);

		for (int i = 9; i >= 0; i--) {
			dinvShiftRows(state);
			dinvSubBytes(state);
			daddRoundKey(i, state, roundKey);
		}
		for (int i = 0; i < 4; i++)
			for (int j = 0; j < 4; j++)
				encryptedPartOfData[4 * i + j] = state[j][i];

		for (int i = 0; i < 16; i++)
			dataOut[16 * index + i] = encryptedPartOfData[i];
	}

	for (int i = 0; i < 4; i++)
	{
		delete[]state[i];
	}
	delete[] state;

}

__host__ void dkeyExpansion(unsigned char *roundKey, unsigned char key[]) {

	int rcon[10] = { 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

	unsigned char temp[4], k;
	for (int i = 0; i < 4; i++) {
		roundKey[i * 4] = key[i * 4];
		roundKey[i * 4 + 1] = key[i * 4 + 1];
		roundKey[i * 4 + 2] = key[i * 4 + 2];
		roundKey[i * 4 + 3] = key[i * 4 + 3];
	}
	for (int i = 0; i < 40; i++) {
		for (int j = 0; j < 4; j++) {
			temp[j] = roundKey[i * 4 + j + 16];
		}

		if (i % 4 == 0) {

			k = temp[0];
			temp[0] = temp[1];
			temp[1] = temp[2];
			temp[2] = temp[3];
			temp[3] = k;

			temp[0] = SBoxValue(temp[0]);
			temp[1] = SBoxValue(temp[1]);
			temp[2] = SBoxValue(temp[2]);
			temp[3] = SBoxValue(temp[3]);

			temp[0] = temp[0] ^ rcon[i / 4];

			roundKey[i * 4 + 16] = roundKey[i * 4 + 0] ^ temp[0];
			roundKey[i * 4 + 17] = roundKey[i * 4 + 1] ^ temp[1];
			roundKey[i * 4 + 18] = roundKey[i * 4 + 2] ^ temp[2];
			roundKey[i * 4 + 19] = roundKey[i * 4 + 3] ^ temp[3];
		}
		else {
			roundKey[i * 4 + 16] = roundKey[i * 4 + 12] ^ roundKey[i * 4];
			roundKey[i * 4 + 17] = roundKey[i * 4 + 13] ^ roundKey[i * 4 + 1];
			roundKey[i * 4 + 18] = roundKey[i * 4 + 14] ^ roundKey[i * 4 + 2];
			roundKey[i * 4 + 19] = roundKey[i * 4 + 15] ^ roundKey[i * 4 + 3];
		}
	}
}

unsigned char* CudaEncryption(unsigned char *data, int width, int height, int whatDoWeDonext) {

	int dataSize = 3 * width*height;
	int numberOfIterations = dataSize / 16;

	unsigned char* roundKey = new unsigned char[176];
	unsigned char key[16] = { 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15 };
	unsigned char *dataIn, *droundKey, *dataOut;
	unsigned char *d_dataIn, *d_dataOut, *d_roundKey;
	int *size, *d_size;
	int keySize = 176;
	dkeyExpansion(roundKey, key);

	size = (int*)malloc(sizeof(int));
	dataIn = (unsigned char*)malloc((dataSize) * sizeof(unsigned char));
	dataOut = (unsigned char*)malloc((dataSize) * sizeof(unsigned char));
	d_dataIn = (unsigned char*)malloc((dataSize) * sizeof(unsigned char));
	d_dataOut = (unsigned char*)malloc((dataSize) * sizeof(unsigned char));
	d_roundKey = (unsigned char*)malloc((keySize) * sizeof(unsigned char));

	dataIn = data;
	*size = dataSize;
	droundKey = roundKey;

	hipMalloc(&d_dataIn, dataSize * sizeof(unsigned char));
	hipMalloc(&d_dataOut, dataSize * sizeof(unsigned char));
	hipMalloc(&d_roundKey, keySize * sizeof(unsigned char));
	hipMalloc(&d_size, sizeof(int));

	hipMemcpy(d_dataIn, dataIn, dataSize * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_roundKey, droundKey, keySize * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_size, size, sizeof(int), hipMemcpyHostToDevice);
	
	cout << "Width: "<<  width << " Height: " << height << endl;

	if (whatDoWeDonext == 1) {
		cout << "Trwa kodowanie na GPU." << endl;
		clock_t t;
		t = clock();
		dencryption << < (numberOfIterations+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_dataIn, d_dataOut, d_size, d_roundKey);
		hipMemcpy(dataOut, d_dataOut, dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
		t = clock() - t;
		cout << "Kodowanie trwalo " << float(t) / CLOCKS_PER_SEC << " sekund." << endl;
		return dataOut;
	}
	else {
		cout << "Trwa dekodowanie na GPU." << endl;
		clock_t t;
		t = clock();
		ddecryption << <(numberOfIterations + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_dataIn, d_dataOut, d_size, d_roundKey);
		hipMemcpy(dataOut, d_dataOut, dataSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
		t = clock() - t;
		cout << "Dekodowanie trwalo " << float(t) / CLOCKS_PER_SEC << " sekund." << endl;
		return dataOut;
	}
}

void encryptionRound(unsigned char **state, unsigned char *roundKey, unsigned char *encryptedPartOfData) {

	addRoundKey(0, state, roundKey);

	for (int i = 1; i <= 10; i++) {
		subBytes(state);
		shiftRows(state);
		addRoundKey(i, state, roundKey);
	}
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < 4; j++)
			encryptedPartOfData[4 * i + j] = state[j][i];

}

void decryptionRound(unsigned char **state, unsigned char *roundKey, unsigned char *encryptedPartOfData) {

	addRoundKey(10, state, roundKey);

	for (int i = 9; i >= 0; i--) {
		invShiftRows(state);
		invSubBytes(state);
		addRoundKey(i, state, roundKey);
	}
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < 4; j++)
			encryptedPartOfData[4 * i + j] = state[j][i];

}

void encryption(unsigned char data[], int width, int height, unsigned char *encryptedData, int whatDoWeDonext) {

	unsigned char input[16];
	unsigned char **state = new unsigned char*[4];
	for (int i = 0; i < 4; i++)
		state[i] = new unsigned char[4];

	int numberOfIterations = width * height * 3 / 16;
	unsigned char* roundKey = new unsigned char[176];
	unsigned char* encryptedPartOfData = new unsigned char[16];
	unsigned char key[16] = { 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15 };

	keyExpansion(roundKey, key);

	cout << "Width: " << width << " Height: " << height << endl;
	cout << "Trwa kodowanie na CPU." << endl;
	clock_t t;
	t = clock();
	for (int j = 0; j < numberOfIterations; j++) {

		for (int i = 0; i < 16; i++) {
			input[i] = data[16 * j + i];
		}

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				state[j][i] = input[4 * i + j];
			}
		}
		if (whatDoWeDonext == 1)
			encryptionRound(state, roundKey, encryptedPartOfData);
		else
			decryptionRound(state, roundKey, encryptedPartOfData);

		for (int i = 0; i < 16; i++) {
			encryptedData[16 * j + i] = encryptedPartOfData[i];
		}
	}

	t = clock() - t;

	if (whatDoWeDonext == 1 )
		cout << "Kodowanie trwalo " << float(t) / CLOCKS_PER_SEC << " sekund." << endl;
	else if(whatDoWeDonext ==2)
		cout << "Dekodowanie trwalo " << float(t) / CLOCKS_PER_SEC << " sekund." << endl;

}

unsigned char* loadImage(unsigned char* entireData, unsigned char* info, string name) {

	FILE * filepoint;
	errno_t err;

	if ((err = fopen_s(&filepoint, name.c_str(), "rb+")) != 0) {
		cout << "Argument Exception" << endl;
		return 0;
	}

	else {

		fread(info, sizeof(unsigned char), 54, filepoint);
		int width = *(int*)&info[18];
		int height = *(int*)&info[22];
		int row_padded = (width * 3 + 3) & (~3);
		unsigned char* data = new unsigned char[row_padded];
		entireData = new unsigned char[width*height * 3];

		for (int i = 0; i < height; i++) {
			fread(data, sizeof(unsigned char), row_padded, filepoint);
			for (int j = 0; j < width * 3; j += 3) {
				//cout << "R: " << (int)data[j] << " G: " << (int)data[j + 1] << " B: " << (int)data[j + 2] << endl;
				entireData[3 * i*width + j] = data[j];
				entireData[3 * i*width + j + 1] = data[j + 1];
				entireData[3 * i*width + j + 2] = data[j + 2];
				//cout << 3 * i*width + j + 2 << endl;

			}
		}
		fclose(filepoint);

		//cout << " wyposuje data" << endl;
		//for (int i = 0; i < width*height * 3 ; i ++)
		//cout << i << " " << (int)entireData[i] << endl;
		return entireData;
	}

}

int *getSizeOfImage(string name) {
	FILE * filepoint;
	errno_t err;

	if ((err = fopen_s(&filepoint, name.c_str(), "rb+")) != 0) {
		cout << "Argument Exception" << endl;
		return 0;
	}

	else {
		unsigned char info[54];
		fread(info, sizeof(unsigned char), 54, filepoint);
		int width = *(int*)&info[18];
		int height = *(int*)&info[22];
		int sizeArray[2];
		sizeArray[0] = width;
		sizeArray[1] = height;
		fclose(filepoint);
		return sizeArray;
	}
}

int main() {

	cout << "Program do zakodowania i odkodowania zdjecia za pomoca algorytmu AES." << endl;
	string input;
	string outputGPU;
	string outputCPU;
	unsigned char * info = new unsigned char[54];

	cout << "Wcisnij 1, aby zakodowac zdjecie." << endl;
	cout << "Wcisnij 2, aby odkodowac zdjecie." << endl;
	cout << "Wcisnij 3, aby opuscic program" << endl;

	int whatDoWeDoNext;
	cin >> whatDoWeDoNext;

	while (whatDoWeDoNext != 3) {

		cout << "Wprowadz nazwe pliku." << endl;
		cin >> input;
		outputCPU = input;
		outputGPU = input;
		input.append(".bmp");
		outputGPU.append("GPU");
		int *vectorSize = getSizeOfImage(input);
		int width = vectorSize[0];
		int height = vectorSize[1];
		int size = width * height * 3;
		unsigned char * entireData = new unsigned char[size];
		unsigned char * encryptedData = new unsigned char[size];
		unsigned char * encryptedDataCuda = new unsigned char[size];
		entireData = loadImage(entireData, info, input);

		if (whatDoWeDoNext == 1 || whatDoWeDoNext == 2) {
			encryption(entireData, width, height, encryptedData, whatDoWeDoNext);
			encryptedDataCuda = CudaEncryption(entireData, width, height, whatDoWeDoNext);
		}
		else
			exit(0);

		if (whatDoWeDoNext == 1) {
			outputGPU.append("_enc.bmp");
			outputCPU.append("_enc.bmp");
		}
		else if (whatDoWeDoNext == 2) {
			outputGPU.append("_decrypted.bmp");
			outputCPU.append("_decrypted.bmp");
		}
		ofstream outputImageGPU(outputGPU, ios::binary);

		for (int i = 0; i < 54; i++)
			outputImageGPU << info[i];

		for (auto i = 0; i <= size - 1; i += 1)
			outputImageGPU << encryptedDataCuda[i];

		outputImageGPU.close();

		ofstream outputImageCPU(outputCPU, ios::binary);

		for (int i = 0; i < 54; i++)
			outputImageCPU << info[i];

		for (auto i = 0; i <= size - 1; i += 1)
			outputImageCPU << encryptedData[i];

		outputImageCPU.close();

		cout << "Wcisnij 1, aby zakodowac zdjecie." << endl;
		cout << "Wcisnij 2, aby odkodowac zdjecie." << endl;
		cout << "Wcisnij 3, aby opuscic program" << endl;
		cin >> whatDoWeDoNext;
	}

	return 0;
}
